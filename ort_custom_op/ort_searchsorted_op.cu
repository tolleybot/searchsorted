#include "hip/hip_runtime.h"

#include "ort_searchsorted_op.h"
#include "searchsorted_cuda_kernel.h"


// Implementation of the Compute function for SearchSortedKernel
template <typename T>
void SearchSortedKernel<T>::Compute(OrtKernelContext* context) {

    Ort::KernelContext ctx(context);
    auto v_a = ctx.GetInput(0);
    auto v_v = ctx.GetInput(1);
    auto v_side_left = ctx.GetInput(2); 
   
    const T* a_data = v_a.GetTensorData<T>();
    const T* v_data = v_v.GetTensorData<T>();
    const bool* side_left_data = v_side_left.GetTensorData<bool>();
    
    auto dimensions_a = v_a.GetTensorTypeAndShapeInfo().GetShape();
    auto dimensions_v = v_v.GetTensorTypeAndShapeInfo().GetShape();
    
    // Prepare the output tensor    
    auto nrow_a = dimensions_a[0];
    auto ncol_a = dimensions_a[1];
    auto nrow_v = dimensions_v[0];
    auto ncol_v = dimensions_v[1];
    auto nrow_res = std::max(nrow_a, nrow_v);

    std::vector<int64_t> output_dims = {nrow_res, ncol_v};
    auto output_tensor = ctx.GetOutput(0, output_dims);

    int64_t* output_data = output_tensor.GetTensorMutableData<int64_t>();

    // Configure kernel dimensions as in PyTorch
    dim3 threads(ncol_v, nrow_res);
    dim3 blocks(1, 1);
    if (nrow_res * ncol_v > 1024) {
        threads.x = static_cast<int>(fmin(1024.0, static_cast<double>(ncol_v)));
        threads.y = static_cast<int>(floor(1024 / threads.x));
        blocks.x = static_cast<int>(ceil(static_cast<double>(ncol_v) / threads.x));
        blocks.y = static_cast<int>(ceil(static_cast<double>(nrow_res) / threads.y));
    }

    // Launch the CUDA kernel
    bool side_left_bool;
    hipMemcpy(&side_left_bool, side_left_data, sizeof(bool), hipMemcpyDeviceToHost);

   // bool side_left_bool = *side_left_data; // Simplification: assuming side_left is a single bool value
    searchsorted_kernel_float(output_data, a_data, v_data, nrow_res, nrow_a, nrow_v, ncol_a, ncol_v, side_left_bool, blocks, threads);
        // Optional: Check for errors after kernel launch
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        // Handle error
        std::cerr << "CUDA error in kernel launch: " << hipGetErrorString(error) << std::endl;
    }

    // Optional: Synchronize device to wait for kernel completion and catch errors
    error = hipDeviceSynchronize();
    if (error != hipSuccess) {
        // Handle error
        std::cerr << "CUDA error on synchronize: " << hipGetErrorString(error) << std::endl;
    }

   

}

template class SearchSortedKernel<float>;
